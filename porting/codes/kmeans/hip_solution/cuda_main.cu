/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
/*   File:         seq_main.c   (an sequential version)                      */
/*   Description:  This program shows an example on how to call a subroutine */
/*                 that implements a simple k-means clustering algorithm     */
/*                 based on Euclid distance.                                 */
/*   Input file format:                                                      */
/*                 ascii  file: each line contains 1 data object             */
/*                 binary file: first 4-byte integer is the number of data   */
/*                 objects and 2nd integer is the no. of features (or        */
/*                 coordinates) of each object                               */
/*                                                                           */
/*   Author:  Wei-keng Liao                                                  */
/*            ECE Department Northwestern University                         */
/*            email: wkliao@ece.northwestern.edu                             */
/*   Copyright, 2005, Wei-keng Liao                                          */
/*                                                                           */
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

// Copyright (c) 2005 Wei-keng Liao
// Copyright (c) 2011 Serban Giuroiu
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.

// -----------------------------------------------------------------------------

#include <stdio.h>
#include <stdlib.h>
#include <string.h>     /* strtok() */
#include <sys/types.h>  /* open() */
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>     /* getopt() */

int      _debug;
#include "kmeans.h"

/*---< usage() >------------------------------------------------------------*/
static void usage(char *argv0, float threshold) {
    char *help =
        "Usage: %s [switches] -i filename -n num_clusters\n"
        "       -i filename    : file containing data to be clustered\n"
        "       -b             : input file is in binary format (default no)\n"
        "       -n num_clusters: number of clusters (K must > 1)\n"
        "       -t threshold   : threshold value (default %.4f)\n"
        "       -o             : output timing results (default no)\n"
        "       -d             : enable debug mode\n";
    fprintf(stderr, help, argv0, threshold);
    exit(-1);
}

/*---< main() >-------------------------------------------------------------*/
int main(int argc, char **argv) {
           int     opt;
    extern char   *optarg;
    extern int     optind;
           int     isBinaryFile, is_output_timing;

           int     numClusters, numCoords, numObjs;
           int    *membership;    /* [numObjs] */
           char   *filename;
           float **objects;       /* [numObjs][numCoords] data objects */
           float **clusters;      /* [numClusters][numCoords] cluster center */
           float   threshold;
           double  timing, io_timing, clustering_timing;
           int     loop_iterations;

    /* some default values */
    _debug           = 0;
    threshold        = 0.001;
    numClusters      = 0;
    isBinaryFile     = 0;
    is_output_timing = 0;
    filename         = NULL;

    while ( (opt=getopt(argc,argv,"p:i:n:t:abdo"))!= EOF) {
        switch (opt) {
            case 'i': filename=optarg;
                      break;
            case 'b': isBinaryFile = 1;
                      break;
            case 't': threshold=atof(optarg);
                      break;
            case 'n': numClusters = atoi(optarg);
                      break;
            case 'o': is_output_timing = 1;
                      break;
            case 'd': _debug = 1;
                      break;
            case '?': usage(argv[0], threshold);
                      break;
            default: usage(argv[0], threshold);
                      break;
        }
    }

    if (filename == 0 || numClusters <= 1) usage(argv[0], threshold);

    if (is_output_timing) io_timing = wtime();

    /* read data points from file ------------------------------------------*/
    objects = file_read(isBinaryFile, filename, &numObjs, &numCoords);
    if (objects == NULL) exit(1);

    if (is_output_timing) {
        timing            = wtime();
        io_timing         = timing - io_timing;
        clustering_timing = timing;
    }

    /* start the timer for the core computation -----------------------------*/
    /* membership: the cluster id for each data object */
    membership = (int*) malloc(numObjs * sizeof(int));
    assert(membership != NULL);

    clusters = cuda_kmeans(objects, numCoords, numObjs, numClusters, threshold,
                          membership, &loop_iterations);

    free(objects[0]);
    free(objects);

    if (is_output_timing) {
        timing            = wtime();
        clustering_timing = timing - clustering_timing;
    }

    /* output: the coordinates of the cluster centres ----------------------*/
    file_write(filename, numClusters, numObjs, numCoords, clusters,
               membership);

    free(membership);
    free(clusters[0]);
    free(clusters);

    /*---- output performance numbers ---------------------------------------*/
    if (is_output_timing) {
        io_timing += wtime() - timing;
        printf("\nPerforming **** Regular Kmeans (CUDA version) ****\n");

        printf("Input file:     %s\n", filename);
        printf("numObjs       = %d\n", numObjs);
        printf("numCoords     = %d\n", numCoords);
        printf("numClusters   = %d\n", numClusters);
        printf("threshold     = %.4f\n", threshold);

        printf("Loop iterations    = %d\n", loop_iterations);

        printf("I/O time           = %10.4f sec\n", io_timing);
        printf("Computation timing = %10.4f sec\n", clustering_timing);
    }

    return(0);
}

