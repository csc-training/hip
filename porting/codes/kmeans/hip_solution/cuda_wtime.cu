/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
/*   File:         wtime.c                                                   */
/*   Description:  a timer that reports the current wall time                */
/*                                                                           */
/*   Author:  Wei-keng Liao                                                  */
/*            ECE Department Northwestern University                         */
/*            email: wkliao@ece.northwestern.edu                             */
/*   Copyright, 2005, Wei-keng Liao                                          */
/*                                                                           */
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */


#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>

double wtime(void) 
{
    double          now_time;
    struct timeval  etstart;
    struct timezone tzp;

    if (gettimeofday(&etstart, &tzp) == -1)
        perror("Error: calling gettimeofday() not successful.\n");

    now_time = ((double)etstart.tv_sec) +              /* in seconds */
               ((double)etstart.tv_usec) / 1000000.0;  /* in microseconds */
    return now_time;
}

#ifdef _TESTING_
int main(int argc, char **argv) {
    double time;

    time = wtime();
    printf("time of day = %10.4f\n", time);

    return 0;
}
#endif

