
#include <hip/hip_runtime.h>
#include <iostream>
#include "hipblas.h"
using namespace std;

const int N = 1 << 30;

int main(){
	float *a_h, *b_h;
	a_h = new float[N];
	b_h = new float[N];
	float *a_d, *b_d;
	for(int i = 0; i < N; i++){
		a_h[i] = 1.0f;
                b_h[i] = 2.0f ;
	}
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipMalloc((void**) &a_d, sizeof(float) * N);
	hipMalloc((void**) &b_d, sizeof(float) * N);
	hipblasSetVector( N, sizeof(float), a_h, 1, a_d, 1);
	hipblasSetVector( N, sizeof(float), b_h, 1, b_d, 1);
	const float s = 2.0f;
	hipblasSaxpy( handle, N, &s, a_d, 1, b_d, 1);
	hipblasGetVector( N, sizeof(float), b_d, 1, b_h, 1);
	hipFree(a_d);
	hipFree(b_d);
	hipblasDestroy(handle);
        float maxError = 0.0f;

	for(int i = 0; i < N; i++)
		maxError = max(maxError, abs(b_h[i]-4.0f));

        cout << "Max error: " << maxError << endl;


	delete[] a_h;
	delete[] b_h;
	return 0;
}
